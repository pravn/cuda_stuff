#include <iostream>
#include <hip/hip_runtime.h>


//find number of points above a certain threshold
//using ballot to set bits 
//then reduce in warp using popc
//reduce warp sums to block level sum using standard parallel reduction in shared memory

//ballot and popc borrowed from the approach in 
//Shane Cook's CUDA Programming book
//single block 

//pretty trivial setting 
#define NUM_WARPS_PER_BLOCK 2
#define NUM_BLOCKS 1
#define NUM_THREADS_PER_BLOCK 64

__global__ void reducer(int *data, int *count ){
  uint tid = blockIdx.x*blockDim.x + threadIdx.x;
  uint warp_id = threadIdx.x >> 5;
  int threshold = 2;

  uint lane_id = threadIdx.x% 32;

  uint warp_set_bits = 0;

  __shared__ uint warp_reduced_count[NUM_WARPS_PER_BLOCK];


  warp_set_bits = __ballot(data[tid] > threshold);

  if(lane_id==0){
    warp_reduced_count[warp_id] = __popc(warp_set_bits);
  }

  __syncthreads();
  //reduce to single value 
  if(warp_id==0){
    for(int i = NUM_WARPS_PER_BLOCK/2; i>0; i>>=1){
      if(tid<i)	warp_reduced_count[tid] += warp_reduced_count[tid+i];
      __syncthreads();
    }
  }

  __syncthreads();
      
  
  if(threadIdx.x==0){
    *count += warp_reduced_count[0];
    printf("reduced count %d\n", *count);
  }

  
}  



int main(){
    int *h_data; 
    int *d_data;
  
    int h_count = 0;
    int *d_count;


  
  h_data = new int[NUM_THREADS_PER_BLOCK];
  hipMalloc((void **) &d_data, sizeof(int)*NUM_THREADS_PER_BLOCK);

  for(int i=0; i<NUM_THREADS_PER_BLOCK; i++){
    h_data[i] = i;
    //    std::cout << h_data[i] << std::endl;
  }


  hipMemcpy(d_data, h_data, NUM_THREADS_PER_BLOCK*sizeof(int), hipMemcpyHostToDevice);

  hipMalloc((void **) &d_count, sizeof(int));
  hipMemcpy(d_count, &h_count, sizeof(int), hipMemcpyHostToDevice);

  
  reducer<<<1, NUM_THREADS_PER_BLOCK>>> (d_data, d_count);

  hipMemcpy(&h_count, d_count, sizeof(int), hipMemcpyDeviceToHost);

  std::cout << "(count > 0) " << h_count << std::endl;
  

  //cleanupxo
  delete[] h_data;
  hipFree(d_data);
  hipFree(d_count);

}
